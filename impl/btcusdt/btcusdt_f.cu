#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * y, float * p1p0,
	uint * ts__d,
	uint P, uint T)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	uint p      = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (t < GRAND_T && mega_t < MEGA_T && p < P) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		assert(t_btcusdt < T);
		//
		float _y = y[ty*P + p];
		//
		assert(_y >= -1 && _y <= +1);
		atomicAdd(&somme_score[0], S(_y, p1p0[t_btcusdt*P + p]));
		assert(S(_y, p1p0[t_btcusdt*P + p]) >= 0);
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint P = btcusdt->Y;
	//
	float * somme__d = cudalloc<float>(1);
	//
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8), KERD(P, 4)), dim3(16,8,4)>>>(
		somme__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		P, btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(P * GRAND_T * MEGA_T));
	//
	cudafree<float>(somme__d);
	    free       (somme   );
	//
	return score;
};