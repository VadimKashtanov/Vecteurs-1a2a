#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

static __global__ void k__df_btcusdt(
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint P)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	uint p      = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (t < GRAND_T && mega_t < MEGA_T && p < P) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		//
		float _y = y[ty*P + p];
		//
		assert(_y >= -1 && _y <= +1);
		//
		atomicAdd(&dy[ty*P + p], dS(_y, p1p0[t_btcusdt*P + p]) / (float)(P * MEGA_T * GRAND_T));
	}
};

void df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint P = btcusdt->Y;
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8), KERD(P, 4)), dim3(16,8,4)>>>(
		y__d, btcusdt->sorties__d, dy__d,
		ts__d,
		P
	);
	ATTENDRE_CUDA();
};